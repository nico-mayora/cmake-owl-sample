#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

using namespace owl;

#define SAMPLES_PER_PIXEL 24
#define MAX_RAY_BOUNCES 50

inline __device__
vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd) {
  vec3f attenuation = 1.f;

  for (int depth=0;depth<MAX_RAY_BOUNCES;depth++) {
    traceRay(self.world, ray,prd);

    // ray didn't hit anything
    if (prd.out.scatterEvent == Missed)
        return attenuation * prd.out.attenuation;

    // ray got absorbed
    if (prd.out.scatterEvent == Absorbed)
      return vec3f(0.f);

    // ray bounced
    attenuation *= prd.out.attenuation;
    ray = Ray(prd.out.scattered_origin, prd.out.scattered_direction, 1e-3f, 1e10f);
  }

  return vec3f(0.f);
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);

  vec3f color = 0.f;
  for (int sampleID = 0; sampleID < SAMPLES_PER_PIXEL; sampleID++) {
    Ray ray;

    const vec2f pixelSample(prd.random(),prd.random());
    const vec2f screen
      = (vec2f(pixelID)+pixelSample)
      / vec2f(self.fbSize);
    const vec3f origin = self.camera.origin;

    const vec3f direction
      = self.camera.lower_left_corner
      + screen.u * self.camera.horizontal
      + screen.v * self.camera.vertical
      - self.camera.origin;

    ray.origin = origin;
    ray.direction = direction;

    color += tracePath(self, ray, prd);
  }

  const int fbOfs = pixelID.x + self.fbSize.x*pixelID.y;

  self.fbPtr[fbOfs]
    = make_rgba(color / (1.f /SAMPLES_PER_PIXEL));
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  auto &prd = owl::getPRD<PerRayData>();

  const auto self = owl::getProgramData<TrianglesGeomData>();

  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Ng     = normalize(cross(B-A,C-A));

  const vec3f rayDir = optixGetWorldRayDirection();
  const auto &material = *self.material;

  prd.out.attenuation = (.2f + .8f*fabs(dot(rayDir,Ng))) * material.albedo;
}

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();

  PerRayData &prd = owl::getPRD<PerRayData>();
  prd.out.scatterEvent = Missed;
  prd.out.attenuation = self.sky_color;
}

