#include "hip/hip_runtime.h"
#include "deviceCode.h"
#include "shading.h"

#include "../../common/cuda/helpers.h"

#include <optix_device.h>
#include "owl/RayGen.h"
#include <cukd/knn.h>

#define CONSTANT_LIGHT_FACTOR 0.1f

using namespace owl;

inline __device__
vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd, int depth) {
  if (!depth) return 0.f;

  uint32_t p0, p1;
  packPointer(&prd, p0, p1);
  optixTrace(self.world,
    ray.origin,
    ray.direction,
    ray.tmin,
    ray.tmax,
    0.f,
    OptixVisibilityMask(255),
    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
    0,
    2,
    0,
    p0, p1
  );
  if (prd.ray_missed)
    return prd.colour;

  auto albedo = prd.hit_record.material.albedo;
  auto diffuse_brdf = prd.hit_record.material.diffuse / PI;

  // Direct light
  vec3f direct_illumination = 0.f;
  for (int l = 0; l < self.numLights; l++) {
    auto current_light = self.lights[l];

    auto shadow_ray_org = prd.hit_record.hitpoint;
    auto light_dir = current_light.pos - shadow_ray_org;
    auto distance_to_light = norm(light_dir);
    light_dir = normalize(light_dir);

    auto light_dot_norm = dot(light_dir, prd.hit_record.normal_at_hitpoint);
    if (light_dot_norm < 0.f) continue; // light hits "behind" triangle

    vec3f light_visibility = 0.f;
    uint32_t u0, u1;
    packPointer(&light_visibility, u0, u1);
    optixTrace(
      self.world,
      shadow_ray_org,
      light_dir,
      EPS,
      distance_to_light * (1.f - EPS),
      0.f,
      OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT
      | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
      | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
      1,
      2,
      1,
      u0, u1
    );

    auto specular_brdf = specularBrdf(prd.hit_record.material.specular,
      light_dir,
      ray.direction,
      prd.hit_record.normal_at_hitpoint);

    direct_illumination += light_visibility
      * CONSTANT_LIGHT_FACTOR
      * static_cast<float>(current_light.power)
      * light_dot_norm
      * (1.f / distance_to_light * distance_to_light)
      * (diffuse_brdf + specular_brdf)
      * current_light.rgb;
  }
  auto direct_term =  albedo * direct_illumination;

  // Specular Reflection
  bool absorbed;
  float coefficient;
  auto out_dir = reflect_or_refract_ray(
    prd.hit_record.material, ray.direction,
    prd.hit_record.normal_at_hitpoint, prd.random,
    absorbed, coefficient
  );

  vec3f specular_term = 0.f;
  // TODO: Structure this properly!
  if (absorbed) goto after_specular;

  auto out_ray = Ray(prd.hit_record.hitpoint, out_dir, EPS, INFTY);

  auto reflected_irradiance = tracePath(self, out_ray, prd, depth-1);
  specular_term = reflected_irradiance * coefficient;

  after_specular:

  return specular_term;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);

  if (pixelID.x == 400 && pixelID.y == 40)
  {
    prd.debug = true;
  }

  auto final_colour = vec3f(0.f);
  for (int sample = 0; sample < self.samples_per_pixel; sample++) {
    const auto random_eps = vec2f(prd.random(), prd.random());
    const vec2f screen = (vec2f(pixelID)+random_eps) / vec2f(self.fbSize);

    Ray ray;
    ray.origin
      = self.camera.pos;
    ray.direction
      = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);

    const auto colour = tracePath(self, ray, prd, self.max_ray_depth);

    final_colour += colour;
  }

  final_colour = final_colour * (1.f / self.samples_per_pixel);

  const int x = pixelID.x;
  const int y = self.fbSize.y - pixelID.y;

  const int fbOfs = x+self.fbSize.x*y;

  self.fbPtr[fbOfs]
    = make_rgba(final_colour);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  auto &prd = owl::getPRD<PerRayData>();
  const auto self = owl::getProgramData<TrianglesGeomData>();

  prd.hit_record.material = *self.material;

  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const auto tmax = optixGetRayTmax();

  prd.hit_record.hitpoint = rayOrg + rayDir * tmax;

  // Calculate normal at hitpoint and flip if it's pointing
  // in the same direction as the incident ray.
  const auto normal = getPrimitiveNormal(self);
  prd.hit_record.normal_at_hitpoint = (dot(rayDir, normal) < 0.f) ? normal : -normal;

  prd.colour = 0.f;
  prd.ray_missed = false;
}

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();

  auto &prd = owl::getPRD<PerRayData>();
  prd.colour = self.sky_colour;
  prd.ray_missed = true;
}

OPTIX_MISS_PROGRAM(shadow)()
{
  // we didn't hit anything, so the light is visible
  vec3f &lightVisbility = getPRD<vec3f>();
  lightVisbility = vec3f(1.f);
}

OPTIX_CLOSEST_HIT_PROGRAM(shadow)() { /* unused */}
OPTIX_ANY_HIT_PROGRAM(shadow)() { /* unused */}
