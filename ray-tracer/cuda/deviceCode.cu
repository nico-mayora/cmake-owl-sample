#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "shading.h"

#include "../../common/cuda/helpers.h"

#include <optix_device.h>
#include "owl/RayGen.h"
#include <cukd/knn.h>

using namespace owl;

inline __device__
vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd) {
  auto acum = vec3f(1.);

  uint32_t p0, p1;
  packPointer( &prd, p0, p1 );
  for (int i = 0; i < self.max_ray_depth; i++) {
    prd.scattered.ray = Ray(0.f, 0.f, EPS, INFTY);

    optixTrace(self.world,
      ray.origin,
      ray.direction,
      ray.tmin,
      ray.tmax,
      0.f,
      OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT,
      0,
      2,
      0,
      p0, p1
    );

    /* trace shadow rays */
    vec3f light_colour = vec3f(0.f);

    const auto lights = self.lights;
    const auto numLights = self.numLights;

    for (int l = 0; l < numLights; l++) {
      auto current_light = lights[l];
      auto shadow_ray_origin = prd.hit_point;
      auto light_direction = current_light.pos - shadow_ray_origin;
      auto distance_to_light = length(light_direction);
      const auto normal = prd.scattered.normal_at_hitpoint;

      auto light_dot_norm = dot(light_direction, normal);
      if (light_dot_norm <= 0.f) continue; // light hits "behind" triangle

      vec3f lightVisibility = 0.f;
      uint32_t u0, u1;
      packPointer(&lightVisibility, u0, u1);
      optixTrace(
        self.world,
        shadow_ray_origin,
        normalize(light_direction),
        EPS,
        distance_to_light * (1.f - EPS),
        0.f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT
        | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
        | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        1,
        2,
        1,
        u0, u1
      );
      light_colour
        += lightVisibility
        * current_light.rgb
        * (light_dot_norm / (distance_to_light * distance_to_light))
        * (static_cast<float>(current_light.power) / numLights)
        * (1.f / self.samples_per_pixel);
    }

    if (isZero(prd.scattered.ray.direction) && isZero(prd.scattered.ray.origin)) {
      return prd.colour;
    }

    ray = prd.scattered.ray;
  }

  return prd.colour;
}

inline __device__
cukd::FixedCandidateList<K_NEAREST_NEIGHBOURS> KNearestPhotons(float3 queryPoint, Photon* photons, int numPoints) {
  cukd::FixedCandidateList<K_NEAREST_NEIGHBOURS> closest(K_MAX_DISTANCE);
  auto sqrDistOfFurthestOneInClosest = cukd::stackBased::knn<
    cukd::FixedCandidateList<K_NEAREST_NEIGHBOURS>,Photon, Photon_traits
  >(
    closest,queryPoint,photons,numPoints
  );
  return closest;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  if (pixelID.x == 0 && pixelID.y == 0){
    for (int i=0; i<5; i++) {
      printf("photon %d: %f %f %f\n", i, self.photons[i].pos.x, self.photons[i].pos.y, self.photons[i].pos.z);
    }
  }

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);
  prd.attenuation = 1.f;

  if (pixelID.x == 0 && pixelID.y == 0) {
    auto queryPoint = make_float3(0.f, 0.f, 0.f);
    auto closest = KNearestPhotons(queryPoint, self.photons, self.numPhotons);

    for (int i = 0; i < K_NEAREST_NEIGHBOURS; i++) {
      auto id = closest.get_pointID(i);
      auto photon = self.photons[id];
      printf("Closest point %d: %f %f %f, %f %f %f\n", i, photon.pos.x, photon.pos.y, photon.pos.z, photon.color.x, photon.color.y, photon.color.z);
    }
  }


  auto final_colour = vec3f(0.f);
  for (int sample = 0; sample < self.samples_per_pixel; sample++) {
    const auto random_eps = vec2f(prd.random(), prd.random());
    const vec2f screen = (vec2f(pixelID)+random_eps) / vec2f(self.fbSize);

    Ray ray;
    ray.origin
      = self.camera.pos;
    ray.direction
      = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);

    const auto colour = tracePath(self, ray, prd);

    final_colour += colour;
  }

  final_colour = final_colour * (1.f / self.samples_per_pixel);

  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;

  self.fbPtr[fbOfs]
    = make_rgba(final_colour);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  auto &prd = owl::getPRD<PerRayData>();
  const auto self = owl::getProgramData<TrianglesGeomData>();
  const auto rgd = owl::getProgramData<RayGenData>();
  const auto material = *self.material;

  if (material.diffuse > 0.f)
    //diffuseAndCausticReflectence(self, prd, rgd);


  // As we can only have one scattered ray, we randomly
  // select either transmission or reflection based on
  // the material's indices.
  if (prd.random() < material.specular / (material.diffuse + material.specular)) {
    specularReflectence(self, prd);
  } else {
    transmissionReflectence(self, prd);
  }
}

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();

  auto &prd = owl::getPRD<PerRayData>();
  prd.colour = self.sky_colour;
}

OPTIX_MISS_PROGRAM(shadow)()
{
    // we didn't hit anything, so the light is visible
    vec3f &prd = getPRD<vec3f>();
    prd = vec3f(1.f);
}

OPTIX_CLOSEST_HIT_PROGRAM(shadow)() { /* unused */}
OPTIX_ANY_HIT_PROGRAM(shadow)() { /* unused */}
