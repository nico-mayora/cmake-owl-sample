#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "../../common/cuda/helpers.h"

#include <optix_device.h>

using namespace owl;

OPTIX_RAYGEN_PROGRAM(pointLightRayGen)(){
  const auto &self = owl::getProgramData<PointLightRGD>();
  const vec2i id = owl::getLaunchIndex();

  PhotonMapperPRD prd;
  prd.random.init(id.x, id.y);
  prd.color = self.color;

  Ray ray;
  ray.origin = self.position;
  ray.direction = randomPointInUnitSphere(prd.random);
  ray.tmin = EPS;

  for(int i = 0; i < self.maxDepth; i++) {
    owl::traceRay(self.world, ray, prd);

    if (i > 0 && prd.event & (ABSORBED | SCATTER_DIFFUSE)) {
      int photonIndex = atomicAdd(self.photonsCount, 1);
      auto photon = &self.photons[photonIndex];
      photon->color = prd.color;
      photon->pos = prd.scattered.origin;
    }

    if (prd.event & (SCATTER_DIFFUSE | SCATTER_SPECULAR | SCATTER_REFRACT)) {
      ray.origin = prd.scattered.origin;
      ray.direction = prd.scattered.direction;
      prd.color = prd.scattered.color;
    } else {
      break;
    }
  }
}

inline __device__ void scatterDiffuse(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_DIFFUSE;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflectDiffuse(normal, prd.random);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

inline __device__ void scatterSpecular(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_SPECULAR;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflect(rayDir, normal);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

inline __device__ void scatterRefract(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_REFRACT;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = refract(rayDir, normal, self.material->refraction_idx);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshClosestHit)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  const auto &self = owl::getProgramData<TrianglesGeomData>();

  const float diffuseProb = self.material->diffuse;
  const float specularProb = self.material->specular + diffuseProb;
  const float transmissionProb = self.material->transmission + specularProb;

  const float randomProb = prd.random();
  if (randomProb < diffuseProb) {
    scatterDiffuse(prd, self);
  } else if (randomProb < diffuseProb + specularProb) {
    scatterSpecular(prd, self);
  } else if (randomProb < transmissionProb) {
    scatterRefract(prd, self);
  } else {
    prd.event = ABSORBED;
  }
}

OPTIX_MISS_PROGRAM(miss)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  prd.event = MISS;
}