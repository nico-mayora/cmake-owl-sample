#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "../../common/cuda/helpers.h"

#include <optix_device.h>

using namespace owl;

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);

  LightSource lightSource;
  auto lightSources = self.lightSources;
  int photon_id = pixelID.x;
  for (int i = 0; i < self.lightsNum; i++) {
    lightSource = lightSources[i];
    if (photon_id < lightSource.num_photons) {
      break;
    } else {
      photon_id -= lightSource.num_photons;
    }
  }

  Ray ray;
  ray.origin = lightSource.pos;
  ray.direction = normalize(randomPointInUnitSphere(prd.random));

  prd.colour = lightSource.rgb;
  for (int i = 0; i < MAX_RAY_BOUNCES; i++) {
    owl::traceRay(self.world, ray, prd);

    if (prd.event == Missed || prd.event == Absorbed) {
      break;
    }

    if (prd.event == ReflectedDiffuse) {
      //int j = atomicAdd(&self.photonsCount, 1);
      int j = pixelID.x;

      self.photons[j].color = prd.colour;
      self.photons[j].pos = prd.scattered.s_origin;
      self.photons[j].dir = prd.scattered.s_direction;
      self.photons[j].is_alive = true;
    }

    ray.origin = prd.scattered.s_origin;
    ray.direction = prd.scattered.s_direction;
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  auto &prd = owl::getPRD<PerRayData>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const auto tmax = optixGetRayTmax();
  const auto &material = *self.material;

  switch (material.surface_type) {
    case LAMBERTIAN: {
      scatterLambertian(prd, self);
      break;
    }
    case SPECULAR: {
      scatterSpecular(prd, self);
      break;
    }
    case GLASS: {
      scatterGlass(prd, self);
      break;
    }
    default: {
      scatterLambertian(prd, self);
      break;
    }
  }
}

OPTIX_MISS_PROGRAM(miss)()
{
  auto &prd = owl::getPRD<PerRayData>();
  prd.event = Missed;
}

