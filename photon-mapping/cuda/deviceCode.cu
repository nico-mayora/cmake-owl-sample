#include "hip/hip_runtime.h"
#include "deviceCode.h"
#include "../../common/cuda/helpers.h"
#define PHOTON_ATTENUATION_FACTOR 150
#define ATTENUATE_PHOTONS false

#include <optix_device.h>

using namespace owl;

inline __device__
vec3f calculateTransmissionDirection(const vec3f &normal, const vec3f &direction, const float refraction_idx, float random) {
  const auto reflected = reflect(normalize(direction), normal);
  vec3f outward_normal;
  vec3f refracted;
  float reflect_prob;
  float cosine;
  float ni_over_nt;

  if (dot(direction, normal) > 0.f) {
    outward_normal = -normal;
    ni_over_nt = refraction_idx;
    cosine = dot(direction, normal);
    cosine = sqrtf(1.f - refraction_idx*refraction_idx*(1.f-cosine*cosine));
  } else {
    outward_normal = normal;
    ni_over_nt = 1.0 / refraction_idx;
    cosine = -dot(direction, normal);// / vec3f(dir).length();
  }

  if (refract(direction, outward_normal, ni_over_nt, refracted))
    reflect_prob = schlickFresnelAprox(cosine, refraction_idx);
  else
    reflect_prob = 1.f;

  if (random < reflect_prob) {
    return reflected;
  } else {
    return refracted;
  }
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);

  LightSource lightSource;
  auto lightSources = self.lightSources;
  int photon_id = pixelID.x;
  for (int i = 0; i < self.lightsNum; i++) {
    lightSource = lightSources[i];
    if (photon_id < lightSource.num_photons) {
      break;
    } else {
      photon_id -= lightSource.num_photons;
    }
  }

  Ray ray;
  ray.origin = lightSource.pos;
  ray.direction = normalize(randomPointInUnitSphere(prd.random));
  prd.colour = lightSource.rgb;

  bool is_alive = true;
  owl::vec3f color = lightSource.rgb;
  for (int i = 0; i < MAX_RAY_BOUNCES; i++) {
//    if (pixelID.x == 0) {
//      printf("i: %d\n", i);
//      printf("ray.origin: %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z);
//      printf("is_alive: %d\n", is_alive);
//    }
    int photon_index = (pixelID.x *  MAX_RAY_BOUNCES) + i;
    Photon photon;
    photon.is_alive = false;

    if (is_alive) {
      owl::traceRay(self.world, ray, prd);

      float russian_roulette = prd.random();

      double d = prd.material.diffuse;
      double s = prd.material.specular;
      double t = prd.material.transmission;

//      if (pixelID.x == 0) {
//        printf("russian_roulette: %f\n", russian_roulette);
//      }
      if (ATTENUATE_PHOTONS && prd.hit_point.distance) {
        color = clampvec(color * PHOTON_ATTENUATION_FACTOR / (prd.hit_point.distance * prd.hit_point.distance), 1);
      }

      if (russian_roulette < d) {
        // Diffuse
        photon.color = color;
        photon.pos = prd.hit_point.origin;
        photon.dir = prd.hit_point.direction;
        photon.is_alive = true;

        auto scatter_direction = prd.hit_point.normal + normalize(randomPointInUnitSphere(prd.random));
        if (dot(scatter_direction, scatter_direction) < EPS) {
          scatter_direction = prd.hit_point.normal;
        }
        ray.origin = prd.hit_point.origin;
        ray.direction = normalize(scatter_direction);
        color *= prd.material.albedo;
      } else if (russian_roulette < d + s) {
        // Specular
        const auto reflected = reflect(normalize(prd.hit_point.direction), prd.hit_point.normal);
        ray.origin = prd.hit_point.origin;
        ray.direction = reflected;
        color *= prd.material.albedo;
      } else if (russian_roulette < d + s + t) {
        // Transmission
        ray.origin = prd.hit_point.origin;
        ray.direction = calculateTransmissionDirection(prd.hit_point.normal, prd.hit_point.direction, prd.material.refraction_idx, prd.random());
        color *= prd.material.albedo;
      } else {
        is_alive = false;
      }
    }
    self.photons[photon_index] = photon;
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  auto &prd = owl::getPRD<PerRayData>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  const auto &material = *self.material;

  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f Ng = getPrimitiveNormal(self);
  const float t = optixGetRayTmax();

  // Copy material to prd
  prd.material.albedo = material.albedo;
  prd.material.diffuse = material.diffuse;
  prd.material.specular = material.specular;
  prd.material.transmission = material.transmission;
  prd.material.refraction_idx = material.refraction_idx;

  // Populate ray data
  prd.hit_point.origin = rayOrg + t * rayDir;
  prd.hit_point.direction = rayDir;
  prd.hit_point.normal = Ng;
  prd.hit_point.distance = norm(t * rayDir);
}

OPTIX_MISS_PROGRAM(miss)()
{
  auto &prd = owl::getPRD<PerRayData>();
  prd.material.diffuse = 0;
  prd.material.specular = 0;
  prd.material.transmission = 0;
}

