#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "../../common/cuda/helpers.h"

#include <optix_device.h>

using namespace owl;

//OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
//{
//  const RayGenData &self = owl::getProgramData<RayGenData>();
//  const vec2i pixelID = owl::getLaunchIndex();
//
//  PerRayData prd;
//  prd.random.init(pixelID.x,pixelID.y);
//
//  LightSource lightSource;
//  auto lightSources = self.lightSources;
//  int photon_id = pixelID.x;
//  for (int i = 0; i < self.lightsNum; i++) {
//    lightSource = lightSources[i];
//    if (photon_id < lightSource.num_photons) {
//      break;
//    } else {
//      photon_id -= lightSource.num_photons;
//    }
//  }
//
//  Ray ray;
//  ray.origin = lightSource.pos;
//  ray.direction = normalize(randomPointInUnitSphere(prd.random));
//  prd.colour = lightSource.rgb;
//
//  bool is_alive = true;
//  owl::vec3f color = lightSource.rgb;
//  for (int i = 0; i < MAX_RAY_BOUNCES; i++) {
//    if (pixelID.x == 0) {
//      //printf("i: %d\n", i);
//      //printf("ray.origin: %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z);
//      //printf("is_alive: %d\n", is_alive);
//    }
//    int photon_index = atomicAdd(self.photonsCount, 1);
//    Photon photon;
//    photon.is_alive = false;
//
//    if (is_alive) {
//      owl::traceRay(self.world, ray, prd);
//
//      if (prd.event == Missed || prd.event == Absorbed) {
//        is_alive = false;
//        if (pixelID.x == 0) {
//          //printf("prd event: Missed or Absorbed\n");
//        }
//      }
//
//      if (prd.event == ReflectedDiffuse || prd.event == ReflectedSpecular) {
//
//        if (prd.event == ReflectedDiffuse) {
//          color = prd.colour;
//          photon.color = color;
//          photon.pos = prd.scattered.s_origin;
//          photon.dir = ray.direction;
//          photon.is_alive = true;
//          if (pixelID.x == 0) {
//            //printf("prd event: ReflectedDiffuse, coef: %f\n", prd.material.diffuseCoefficient);
//          }
//        } else {
//          if (pixelID.x == 0) {
//            //printf("prd event: ReflectedSpecular, coef: %f\n", prd.material.reflectivity);
//          }
//        }
//
//        float russian_roulette = prd.random();
//
//        double d = prd.material.diffuseCoefficient;
//        double s = prd.material.reflectivity;
//
//        // Currently objects are either diffuse or specular, and the consequent ray is always stored in prd.scatered
//        // When we support multiple coefs per material, we should check for different rays here
//        if (pixelID.x == 0) {
//          //printf("russian_roulette: %f\n", russian_roulette);
//        }
//        if (russian_roulette < d) {
//          if (pixelID.x == 0) {
//            //printf("russian_roulette < d\n");
//          }
//          ray.origin = prd.scattered.s_origin;
//          ray.direction = prd.scattered.s_direction;
//        } else if (russian_roulette < d + s) {
//          if (pixelID.x == 0) {
//            //printf("russian_roulette < d + s\n");
//          }
//          ray.origin = prd.scattered.s_origin;
//          ray.direction = prd.scattered.s_direction;
//        } else {
//          if (pixelID.x == 0) {
//            //printf("russian_roulette MISS\n");
//          }
//          is_alive = false;
//        }
//      }
//
//      if (prd.event == Refraction) {
//        if (pixelID.x == 0) {
//          //printf("prd event: Refraction\n");
//        }
//        color = prd.colour;
//        ray.origin = prd.scattered.s_origin;
//        ray.direction = prd.scattered.s_direction;
//      }
//    }
//
//    photon.color = vec3f(0.5f, 1.f, 0.2f);
//
//    self.photons[photon_index] = photon;
//  }
//}
//
//OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
//{
//  auto &prd = owl::getPRD<PerRayData>();
//
//  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
//  const auto &material = *self.material;
//
//  switch (material.surface_type) {
//    case LAMBERTIAN: {
//      scatterLambertian(prd, self);
//      break;
//    }
//    case SPECULAR: {
//      scatterSpecular(prd, self);
//      break;
//    }
//    case GLASS: {
//      scatterGlass(prd, self);
//      break;
//    }
//    default: {
//      scatterLambertian(prd, self);
//      break;
//    }
//  }
//}
//
//OPTIX_MISS_PROGRAM(miss)()
//{
//  auto &prd = owl::getPRD<PerRayData>();
//  prd.event = Missed;
//}

OPTIX_RAYGEN_PROGRAM(pointLightRayGen)(){
  const auto &self = owl::getProgramData<PointLightRGD>();
  const vec2i id = owl::getLaunchIndex();

  const double u = (double)id.x / self.dims.x;
  const double v = (double)id.y / self.dims.y;
  const double theta = 2.0 * M_PI * u;
  const double phi = acos(2.0 * v - 1.0);

  PhotonMapperPRD prd;
  prd.random.init(id.x, id.y);
  prd.color = self.color;

  Ray ray;
  ray.origin = self.position;
  ray.direction = randomPointInUnitSphere(prd.random);

  for(int i = 0; i < 100; i++) {
    owl::traceRay(self.world, ray, prd);

    if (prd.event & (ABSORBED | SCATTER_DIFFUSE)) {
      int photonIndex = atomicAdd(self.photonsCount, 1);
      auto photon = &self.photons[photonIndex];
      photon->color = prd.color;
      photon->pos = prd.scattered.origin;
    }

    if (prd.event & (SCATTER_DIFFUSE | SCATTER_SPECULAR | SCATTER_REFRACT)) {
      ray.origin = prd.scattered.origin;
      ray.direction = prd.scattered.direction;
      prd.color = prd.scattered.color;
    } else {
      break;
    }
  }
}

inline __device__ void scatterDiffuse(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_DIFFUSE;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflectDiffuse(normal, prd.random);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

inline __device__ void scatterSpecular(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_SPECULAR;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflect(rayDir, normal);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshClosestHit)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  const auto &self = owl::getProgramData<TrianglesGeomData>();

  const float specularProb = self.material->specular;

  const vec3f albedo = self.material->albedo;
  const float diffuseProb = max(albedo.x, max(albedo.y, albedo.z)) * (1.0f - specularProb);

  const float randomProb = prd.random();
  if (randomProb < diffuseProb) {
    scatterDiffuse(prd, self);
  } else if (randomProb < diffuseProb + specularProb) {
    scatterSpecular(prd, self);
  } else {
    prd.event = ABSORBED;
  }
}

OPTIX_MISS_PROGRAM(miss)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  prd.event = MISS;
}